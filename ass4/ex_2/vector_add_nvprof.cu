#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

typedef float DataType;

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        out[idx] = in1[idx] + in2[idx];
    }
}

int main(int argc, char **argv) {
    int inputLength = (argc > 1) ? atoi(argv[1]) : 1048576;
    int S_seg = inputLength / 4;  // Segment size

    // Allocate pinned memory
    DataType *hostInput1, *hostInput2, *hostOutput;
    hipHostMalloc(&hostInput1, inputLength * sizeof(DataType));
    hipHostMalloc(&hostInput2, inputLength * sizeof(DataType));
    hipHostMalloc(&hostOutput, inputLength * sizeof(DataType));

    // Initialize input vectors
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = rand() / (DataType)RAND_MAX;
        hostInput2[i] = rand() / (DataType)RAND_MAX;
    }

    // Allocate GPU memory
    DataType *deviceInput1, *deviceInput2, *deviceOutput;
    hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

    // Create streams and events
    hipStream_t streams[4];
    hipEvent_t startEvent, stopEvent;
    hipEvent_t copyCompleteEvents[4];
    hipEvent_t kernelCompleteEvents[4];
    
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    
    for (int i = 0; i < 4; i++) {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&copyCompleteEvents[i]);
        hipEventCreate(&kernelCompleteEvents[i]);
    }

    // Record start event
    hipEventRecord(startEvent);

    // Process segments using multiple streams
    const int threadsPerBlock = 256;
    
    for (int offset = 0; offset < inputLength; offset += S_seg) {
        int currentSegSize = min(S_seg, inputLength - offset);
        int streamIdx = (offset/S_seg) % 4;
        int blocksPerGrid = (currentSegSize + threadsPerBlock - 1) / threadsPerBlock;

        // Asynchronous memory copies to device
        hipMemcpyAsync(&deviceInput1[offset], 
                       &hostInput1[offset],
                       currentSegSize * sizeof(DataType), 
                       hipMemcpyHostToDevice, 
                       streams[streamIdx]);
                       
        hipMemcpyAsync(&deviceInput2[offset], 
                       &hostInput2[offset],
                       currentSegSize * sizeof(DataType), 
                       hipMemcpyHostToDevice, 
                       streams[streamIdx]);
                       
        // Record completion of copies
        hipEventRecord(copyCompleteEvents[streamIdx], streams[streamIdx]);

        // Launch kernel
        vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[streamIdx]>>>
              (&deviceInput1[offset], 
               &deviceInput2[offset], 
               &deviceOutput[offset], 
               currentSegSize);
               
        // Record completion of kernel
        hipEventRecord(kernelCompleteEvents[streamIdx], streams[streamIdx]);

        // Asynchronous memory copy back to host
        hipMemcpyAsync(&hostOutput[offset], 
                       &deviceOutput[offset],
                       currentSegSize * sizeof(DataType), 
                       hipMemcpyDeviceToHost, 
                       streams[streamIdx]);
    }

    // Record stop event
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);

    // Calculate and print elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
    printf("Execution time: %f ms\n", milliseconds);

    // Cleanup
    for (int i = 0; i < 4; i++) {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(copyCompleteEvents[i]);
        hipEventDestroy(kernelCompleteEvents[i]);
    }
    
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    return 0;
}